#include "hip/hip_runtime.h"
#include <iostream>

#include <hip/hip_runtime.h>
#include <hiprand.h>

#include <thrust/transform.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

//defines for error checking and handling
#define CUDA_CALL(x) do { hipError_t err = (x); if(err != hipSuccess){ \
    printf("Error %d at: %s:%d\n", err, __FILE__, __LINE__); \
    exit(EXIT_FAILURE);}} while(0)

#define CURAND_CALL(x) do { hiprandStatus_t err = (x); if(err !=HIPRAND_STATUS_SUCCESS) { \
    printf("Error %d at %s:%d\n",err, __FILE__,__LINE__);\
    exit(EXIT_FAILURE);}} while(0)

//constant definitions
enum
{
    dimension = 30,
    experiments = 1000000,
};

#define PI 3.14159265358979323846
#define PI_2 (PI * 2.0)
#define E 2.71828182845904523536
#define ackleys_one_constant (1.0 / pow(E, 0.2))

/** \brief Function that takes a given number and rounds it up to the nearest power of 2
 *
 * @param num number to be rounded
 * @return number rounded to the nearest power of 2
 */
__host__
int power_2_round(int num) {
    // Check if the given number is already a power of 2
    if ((num & (num - 1)) == 0) {
        return num;
    }

    // Find the position of the most significant bit
    int msbPosition = static_cast<int>(log2(num)) + 1;

    // Calculate the nearest power of 2 using the left shift operator
    int nearestPowerOf2 = 1 << msbPosition;

    return nearestPowerOf2;
}

/** \brief Builds a CuRAND generator from some given initial parameters.
 * Useful for configuring random number generators neatly
 * @param out_gen An out parameter that will be the configured generator upon success
 * @param rng_type The RNG type desired for this generator
 * @param seed An integer that seeds the given generator. Default = 0 means a seed will be randomly generated
 * @param offset An offset used to skip portions of the RNG cycle, effectively a random starting point. Default = 0 means an offset will be randomly generated
 */
__host__
void build_curand_generator(hiprandGenerator_t &out_gen, hiprandRngType_t rng_type, int seed = 0, unsigned int offset = 0)
{
    CURAND_CALL(hiprandCreateGenerator(&out_gen, rng_type));

    if(seed == 0) seed = static_cast<int>(time(nullptr));
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(out_gen, seed));

    hiprandOrdering_t order;
    switch(rng_type)
    {
        case HIPRAND_RNG_PSEUDO_XORWOW:
            order = HIPRAND_ORDERING_PSEUDO_SEEDED;
            break;
        case HIPRAND_RNG_PSEUDO_MRG32K3A:
            order = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
            break;
        case HIPRAND_RNG_PSEUDO_MTGP32:
            //order = HIPRAND_ORDERING_PSEUDO_BEST;
            //break;
        case HIPRAND_RNG_PSEUDO_MT19937:
            order = HIPRAND_ORDERING_PSEUDO_BEST;
            break;
        case HIPRAND_RNG_PSEUDO_PHILOX4_32_10:
            order = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
            break;
        default:
            order = HIPRAND_ORDERING_PSEUDO_DEFAULT;
            break;
    }
    CURAND_CALL(hiprandSetGeneratorOrdering(out_gen, order));

    if(rng_type != HIPRAND_RNG_PSEUDO_MTGP32 && rng_type != HIPRAND_RNG_PSEUDO_MT19937) {
        if (offset == 0) {
            unsigned int *random_offset;
            CUDA_CALL(hipMallocManaged(&random_offset, sizeof(unsigned int)));
            CURAND_CALL(hiprandGenerate(out_gen, random_offset, 1));
            CUDA_CALL(hipDeviceSynchronize());
            offset = *random_offset;
            CUDA_CALL(hipFree(random_offset));
        }

        CURAND_CALL(hiprandSetGeneratorOffset(out_gen, offset));
    }
}

//function declarations. implementations are at end of file
typedef void (*EvalFuncPtr)(const float*, float*, const int);
__global__ void eval_schwefel(const float* input, float* ans, int rounded_dimension);
__global__ void eval_dejong(const float* input, float* ans, int rounded_dimension);
__global__ void eval_rosenbrock(const float* input, float* ans, int rounded_dimension);
__global__ void eval_rastrigin(const float* input, float* ans, int rounded_dimension) ;
__global__ void eval_griewangk(const float* input, float* ans, int rounded_dimension);
__global__ void eval_sine_envelope_sine(const float* input, float* ans, int rounded_dimension);
__global__ void eval_stretch_v_sine(const float* input, float* ans, int rounded_dimension);
__global__ void eval_ackley_one(const float* input, float* ans, int rounded_dimension);
__global__ void eval_ackley_two(const float* input, float* ans, int rounded_dimension);
__global__ void eval_egg_holder(const float* input, float* ans, int rounded_dimension);

/**
 * \brief Simple hash function to return the name of a given function
 * @param i Index of the function according to the Canvas assignment's ordering
 * @return Name of the function at index i
 */
__host__ std::string get_func_name(const int i)
{
    switch(i)
    {
        case 0: return "Schwefel";
        case 1: return "De Jong 1";
        case 2: return "Rosenbrock's Saddle";
        case 3: return "Rastrigin";
        case 4: return "Griewangk";
        case 5: return "Sine Envelope Sine Wave";
        case 6: return "Stretch V Sine Wave";
        case 7: return "Ackley One";
        case 8: return "Ackley Two";
        case 9: return "Egg Holder";
        default: return "Undefined Function!!!!!!!";
    }
}

/**
 * \brief Simple hash function to return the name of a given generator
 * @param i Index of the generator according to the ordering of generators array
 * @return Name of the generator at index i
 */
__host__ std::string get_gen_name(const int i)
{
    switch(i)
    {
        case 0: return "XORWow";
        case 1: return "Mersenne Twister";
        case 2: return "Philox";
        default: return "Undefined Generator!!!!!!!";
    }
}


int main() {
    const int rounded_dimension = power_2_round(dimension);
    const int N = rounded_dimension * experiments;

    hiprandGenerator_t gen_xorwow;
    build_curand_generator(gen_xorwow, HIPRAND_RNG_PSEUDO_XORWOW);

    hiprandGenerator_t gen_mt;
    build_curand_generator(gen_mt, HIPRAND_RNG_PSEUDO_MT19937);

    hiprandGenerator_t gen_philox;
    build_curand_generator(gen_philox, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);

    const int num_generators = 3;
    hiprandGenerator_t generators[num_generators] = {gen_xorwow, gen_mt, gen_philox};

    const int num_functions = 10;
    EvalFuncPtr eval_ptrs[num_functions] = {eval_schwefel, eval_dejong, eval_rosenbrock, eval_rastrigin, eval_griewangk,
                                            eval_sine_envelope_sine, eval_stretch_v_sine, eval_ackley_one,
                                            eval_ackley_two, eval_egg_holder};

    int bounds[num_functions] = {512, 100, 100, 30, 500, 30, 30, 32, 32, 500};

    int experiments_per_thread = 1024;
    int num_blocks = (experiments + experiments_per_thread - 1) / experiments_per_thread;

    for (int i = 0; i < num_functions; i++) {
        auto eval_ptr = eval_ptrs[i];
        int bound = bounds[i];

        for(int j = 0; j < num_generators; j++) {
            hiprandGenerator_t gen = generators[j];

            //initialize host and device rng arrays
            float *device_rng_input, *host_rng_input;
            host_rng_input = (float *) calloc(N, sizeof(float));
            hipMalloc((void **) &device_rng_input, N * sizeof(float));

            //generate N random numbers
            CURAND_CALL(hiprandGenerateUniform(gen, device_rng_input, N));
            CUDA_CALL(hipDeviceSynchronize());

            //transform the random numbers to be from (0, 1] to [-bound, bound]
            thrust::device_ptr<float> device_ptr = thrust::device_pointer_cast(device_rng_input);
            thrust::transform(device_ptr, device_ptr + N, device_ptr,
                              [=]__device__(float x) { return (x - 0.5f) * 2.0f * static_cast<double>(bound); });

            //copy the random numbers to host array
            CUDA_CALL(hipMemcpy(host_rng_input, device_rng_input, N * sizeof(float), hipMemcpyDeviceToHost));

            //initialize host and device answer arrays
            float *host_ans, *dev_ans;
            host_ans = (float *) calloc(experiments, sizeof(float));
            hipMalloc((void **) &dev_ans, experiments * sizeof(float));

            //evaluate the function and have host wait
            eval_ptr<<<num_blocks, experiments_per_thread>>>(device_rng_input, dev_ans, rounded_dimension);
            CUDA_CALL(hipDeviceSynchronize());

            //copy answer array from device to host
            hipMemcpy(host_ans, dev_ans, experiments * sizeof(float), hipMemcpyDeviceToHost);

            //perform statistics using the thrust library for vector operations
            thrust::host_vector<float> answers(experiments, 0);
            thrust::copy(host_ans, host_ans + experiments, answers.begin());
            thrust::sort(answers.begin(), answers.end());

            const double best_ans = *thrust::min_element(answers.begin(), answers.end());
            const double worst_ans = *thrust::max_element(answers.begin(), answers.end());
            const double sum_ans = thrust::reduce(answers.begin(), answers.end());

            const double mean = sum_ans / experiments;

            const int midpoint = static_cast<int>(answers.size()) / 2;
            const double median = answers.size() % 2 == 0 ? (answers[midpoint] + answers[midpoint + 1]) /2 : answers[midpoint];

            double sum_dev = 0.0;
            for (const auto &sol: answers) {
                sum_dev += pow(sol - mean, 2);
            }
            const double std_dev = sqrt(sum_dev / experiments);

            //output statistics
            std::cout << "----- Results for " << get_func_name(i) << " with the " << get_gen_name(j) << " PRNG -----" << std::endl;
            std::cout << "Average:            " << mean << std::endl;
            std::cout << "Best answer:        " << best_ans << std::endl;
            std::cout << "Worst answer:       " << worst_ans << std::endl;
            std::cout << "Median:             " << median << std::endl;
            std::cout << "Standard deviation: " << std_dev << std::endl;
            //std::cout << get_func_name(i) << " & " << mean << " & " << std_dev << " & " << best_ans << " & " << worst_ans << " & " << median << " TIME \\\\" << std::endl;
            std::cout << "" << std::endl;

            //always remember to free memory
            free(host_ans);
            free(host_rng_input);
            hipFree(dev_ans);
            hipFree(device_rng_input);
        }
        std::cout << "============================================================" << std::endl;
    }

    CUDA_CALL(hipDeviceReset());
    CURAND_CALL(hiprandDestroyGenerator(gen_xorwow));
    CURAND_CALL(hiprandDestroyGenerator(gen_mt));
    CURAND_CALL(hiprandDestroyGenerator(gen_philox));

    return EXIT_SUCCESS;
}

/*
 * Function implementations
 */

__global__
void eval_schwefel(const float* input, float* ans, const int rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end; i++)
    {
        const double x = input[i];

        sum += -x * sin(sqrt(abs(x)));
    }
    double a = (418.9829 * dimension) - sum;
    ans[index] = static_cast<float>(a);
}

__global__
void eval_dejong(const float* input, float* ans, const int rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end; i++)
    {
        const double x = input[i];

        sum += x * x;
    }
    ans[index] = static_cast<float>(sum);
}

__global__
void eval_rosenbrock(const float* input, float* ans, const int rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;
        const double x_i = input[i + 1];

        sum += 100 * pow(x_2 - x_i, 2) + pow(1 - x, 2);
    }
    ans[index] = static_cast<float>(sum);
}

__global__
void eval_rastrigin(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;

        sum += x_2 - 10 * cos(PI_2 * x);
    }
    double a = (10 * dimension) * sum;
    ans[index] = static_cast<float>(a);
}

__global__
void eval_griewangk(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    double prod = 1.0;
    for(unsigned int i = start; i < end; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;

        sum += x_2 / 4000.0;
        prod *= cos(x / sqrt(i + 1.0));
    }
    double a = 1 + sum - prod;
    ans[index] = static_cast<float>(a);
}

__global__
void eval_sine_envelope_sine(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;
        const double x_i = input[i + 1];
        const double x_ii = x_i * x_i;

        const double numerator = pow(sin(x_2 + x_ii - 0.5), 2);
        const double denominator = pow(1.0 + 0.001 * (x_2 + x_ii), 2);
        sum += 0.5 + numerator / denominator;
    }
    double a = -sum;
    ans[index] = static_cast<float>(a);
}

__global__
void eval_stretch_v_sine(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;
        const double x_i = input[i + 1];
        const double x_ii = x_i * x_i;

        const double left_root = pow(x_2 + x_ii, 0.25);
        const double right_root = pow(x_2 + x_ii, 0.1);

        sum += left_root * pow(50.0 * sin(right_root), 2) + 1.0;
    }
    ans[index] = static_cast<float>(sum);
}

__global__
void eval_ackley_one(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;
        const double x_i = input[i + 1];
        const double x_ii = x_i * x_i;

        const double root = sqrt(x_2 + x_ii);
        const double trig = 3.0 * (cos(2.0 * x) + sin(2.0 * x_i));

        sum += ackleys_one_constant * root + trig;
    }
    ans[index] = static_cast<float>(sum);
}

__global__
void eval_ackley_two(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_2 = x * x;
        const double x_i = input[i + 1];
        const double x_ii = x_i * x_i;

        const double root = pow(E, 0.2 * sqrt((x_2 + x_ii) / 2.0));
        const double trig = pow(E, 0.5 * (cos(PI_2 * x) + cos(PI_2 * x_i)));
        sum += 20.0 + E - 20.0 / root - trig;
    }
    ans[index] = static_cast<float>(sum);
}

__global__
void eval_egg_holder(const float* input, float* ans, const int  rounded_dimension)
{
    const unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
    const unsigned int start = index * rounded_dimension;
    const unsigned int end = start + dimension;

    double sum = 0.0;
    for(unsigned int i = start; i < end - 1; i++)
    {
        const double x = input[i];
        const double x_i = input[i + 1];

        const double left_root = sqrt(abs(x - x_i - 47.0));
        const double right_root = sqrt(abs(x_i + 47.0 + x / 2.0));
        sum += -x * sin(left_root) - (x_i + 47.0) * sin(right_root);
    }
    ans[index] = static_cast<float>(sum);
}