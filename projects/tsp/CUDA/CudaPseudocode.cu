
#include "hip/hip_runtime.h"


#include <stdio.h>


#include <iostream>
#include <string>
#include <string.h>
using namespace std;


void getSTSPAdjacencyMatrix(double* matrix, string location, int problemSize) {}

void getATSPAdjacencyMatrix(int* matrix, string location, int nullKey) {}

void cudaHandleError(hipError_t error) {
    if (error != hipSuccess) {
        cout << "Failed to perform device operation: " << hipGetErrorString(error);
    }
}

int main() {


    // for a given problem size
    int STSPproblemSize = 1400; // number of cities
    int ATSPproblemSize = 65; // number of cites

    // and data in local file at a given location
    string STSPLocation = "fl1400.tsp";
    string ATSPLocation = "ftv64.atsp";

    // for a given number of ants
    int numAnts = 10000;

    // run a given number of iterations
    int numIterations = 1000;

    // and possibly some null key for data integrity
    int nullKey = 100000000;

    // populate an adjacency matrix of the problem
    double* STSPAdjacencyMatrix = (double*)malloc(sizeof(double) * STSPproblemSize * STSPproblemSize);
    int* ATSPAdjacencyMatrix = (int*)malloc(sizeof(int) * ATSPproblemSize * ATSPproblemSize);


    getSTSPAdjacencyMatrix(STSPAdjacencyMatrix, STSPLocation, STSPproblemSize);
    getATSPAdjacencyMatrix(ATSPAdjacencyMatrix, ATSPLocation, nullKey);


    // create coppies of the problems on the device
    double* device_STSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_STSPAdjacencyMatrix, sizeof(double) * STSPproblemSize * STSPproblemSize));
    cudaHandleError(hipMemcpy(device_STSPAdjacencyMatrix, STSPAdjacencyMatrix, sizeof(double) * STSPproblemSize * STSPproblemSize, hipMemcpyHostToDevice));

    int* device_ATSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_ATSPAdjacencyMatrix, sizeof(int) * ATSPproblemSize * ATSPproblemSize));
    cudaHandleError(hipMemcpy(device_ATSPAdjacencyMatrix, ATSPAdjacencyMatrix, sizeof(int) * ATSPproblemSize * ATSPproblemSize, hipMemcpyHostToDevice));


    // allocate pheromone matrix on host and device
    double* STSPpheromoneMatrix = (double*)malloc(sizeof(double) * STSPproblemSize * STSPproblemSize);

    double* device_STSPpheromoneMatrix;
    cudaHandleError(hipMalloc(&device_STSPpheromoneMatrix, sizeof(double) * STSPproblemSize * STSPproblemSize));


    double* ATSPpheromoneMatrix = (double*)malloc(sizeof(double) * ATSPproblemSize * ATSPproblemSize);
    
    double* device_ATSPpheromoneMatrix;
    cudaHandleError(hipMalloc(&device_ATSPpheromoneMatrix, sizeof(double) * ATSPproblemSize * ATSPproblemSize));


    // allocate ant histories on matrix


    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result




    // free all used memory

        // device
    cudaHandleError(hipFree(device_STSPAdjacencyMatrix));
    cudaHandleError(hipFree(device_ATSPAdjacencyMatrix));

    cudaHandleError(hipFree(device_STSPpheromoneMatrix));
    cudaHandleError(hipFree(device_ATSPpheromoneMatrix));


        // host
    free(STSPAdjacencyMatrix);
    free(ATSPAdjacencyMatrix);

    free(STSPpheromoneMatrix);
    free(ATSPpheromoneMatrix);


}