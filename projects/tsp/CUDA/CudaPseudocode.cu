
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


#include <iostream>
#include <string>
#include <string.h>
using namespace std;


void getSTSPAdjacencyMatrix(double* matrix, string location, int problemSize) {}

void getATSPAdjacencyMatrix(int* matrix, string location, int nullKey) {}


void ACOsolveSTSP(int problemSize, string location, int numAnts, int numIterations){
    // populate an adjacency matrix of the problem
    double* adjacencyMatrix = (double*)malloc(sizeof(double) * STSPproblemSize * STSPproblemSize);


    getSTSPAdjacencyMatrix(adjacencyMatrix, location, problemSize);


    // create coppies of the problems on the device
    double* device_adjacencyMatrix;
    cudaHandleError(hipMalloc(&device_adjacencyMatrix, sizeof(double) * problemSize * problemSize));
    cudaHandleError(hipMemcpy(device_adjacencyMatrix, adjacencyMatrix, sizeof(double) * problemSize * problemSize, hipMemcpyHostToDevice));



    // allocate pheromone matrix on host and device
    double* pheromoneMatrix = (double*)malloc(sizeof(double) * problemSize * problemSize);

    double* device_pheromoneMatrix;
    cudaHandleError(hipMalloc(&device_pheromoneMatrix, sizeof(double) * problemSize * problemSize));



    // allocate ant histories on matrix
    int* antHistories = (int*)malloc(sizeof(int) * numAnts * problemSize);

    int* device_antHistories;
    cudaHandleError(hipMalloc(&device_antHistories, sizeof(int) * numAnts * problemSize));

    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result




    // free all used memory

        // device
    cudaHandleError(hipFree(device_adjacencyMatrix));
    cudaHandleError(hipFree(device_pheromoneMatrix));
    cudaHandleError(hipFree(device_antHistories));

        // host
    free(adjacencyMatrix);
    free(pheromoneMatrix);
    free(antHistories);
}

void ACOsolveATSP(int problemSIze, string location, int numAnts, int numIterations, int nullKey){
    // populate an adjacency matrix of the problem
    int* adjacencyMatrix = (int*)malloc(sizeof(int) * problemSize * problemSize);

    getATSPAdjacencyMatrix(adjacencyMatrix, location, nullKey);


    // create coppies of the problems on the device
    int* device_adjacencyMatrix;
    cudaHandleError(hipMalloc(&device_adjacencyMatrix, sizeof(int) * problemSize * problemSize));
    cudaHandleError(hipMemcpy(device_adjacencyMatrix, adjacencyMatrix, sizeof(int) * problemSize * problemSize, hipMemcpyHostToDevice));


    // allocate pheromone matrix on host and device
    double* pheromoneMatrix = (double*)malloc(sizeof(double) * problemSize * problemSize);
    
    double* device_pheromoneMatrix;
    cudaHandleError(hipMalloc(&device_pheromoneMatrix, sizeof(double) * problemSize * problemSize));


    // allocate ant histories on matrix
    int* antHistories = (int*)malloc(sizeof(int) * numAnts * problemSize);

    int* device_antHistories;
    cudaHandleError(hipMalloc(&device_antHistories, sizeof(int) * numAnts * problemSize));

    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result
    



    // free all used memory

        // device
    cudaHandleError(hipFree(device_ATSPAdjacencyMatrix));
    cudaHandleError(hipFree(device_ATSPpheromoneMatrix));
    cudaHandleError(hipFree(device_ATSPAntHistories));

        // host
    free(ATSPAdjacencyMatrix);
    free(ATSPpheromoneMatrix);
    free(ATSPAntHistories);

}

void cudaHandleError(hipError_t error) {
    if (error != hipSuccess) {
        cout << "Failed to perform device operation: " << hipGetErrorString(error);
    }
}

int main() {


    // for a given problem size
    int STSPproblemSize = 1400; // number of cities
    int ATSPproblemSize = 65; // number of cites

    // and data in local file at a given location
    string STSPLocation = "fl1400.tsp";
    string ATSPLocation = "ftv64.atsp";

    // for a given number of ants
    int numAnts = 10000;

    // run a given number of iterations
    int numIterations = 1000;

    // and possibly some null key for data integrity
    int nullKey = 100000000;

    ACOsolveSTSP(STSPproblemSize, STSPLocation, numAnts, numIterations);
    ACOsolveATSP(ATSPproblemSize, ATSPLocation, numAnts, numIterations, nullKey);

}