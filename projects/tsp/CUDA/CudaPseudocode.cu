
#include "hip/hip_runtime.h"


#include <stdio.h>


#include <iostream>
#include <string>
#include <string.h>
using namespace std;


void getSTSPAdjacencyMatrix(double* matrix, string location, int problemSize) {}

void getATSPAdjacencyMatrix(int* matrix, string location, int nullKey) {}

void cudaHandleError(hipError_t error) {
    if (error != hipSuccess) {
        cout << "Failed to perform device operation: " << hipGetErrorString(error);
    }
}

int main() {


    // for a given problem size
    int STSPproblemSize = 1400; // number of cities
    int ATSPproblemSize = 65; // number of cites

    // and data in local file at a given location
    string STSPLocation = "fl1400.tsp";
    string ATSPLocation = "ftv64.atsp";

    // and possibly some null key for data integrity
    int nullKey = 100000000;

    // populate an adjacency matrix of the problem
    double* STSPAdjacencyMatrix = (double*)malloc(sizeof(double) * STSPproblemSize * STSPproblemSize);
    int* ATSPAdjacencyMatrix = (int*)malloc(sizeof(int) * ATSPproblemSize * ATSPproblemSize);


    getSTSPAdjacencyMatrix(STSPAdjacencyMatrix, STSPLocation, STSPproblemSize);
    getATSPAdjacencyMatrix(ATSPAdjacencyMatrix, ATSPLocation, nullKey);


    // create coppies of the problems on the device
    double* device_STSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_STSPAdjacencyMatrix, sizeof(double) * STSPproblemSize * STSPproblemSize));
    cudaHandleError(hipMemcpy(device_STSPAdjacencyMatrix, STSPAdjacencyMatrix, sizeof(double) * STSPproblemSize * STSPproblemSize, hipMemcpyHostToDevice));

    int* device_ATSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_ATSPAdjacencyMatrix, sizeof(int) * ATSPproblemSize * ATSPproblemSize));
    cudaHandleError(hipMemcpy(device_ATSPAdjacencyMatrix, ATSPAdjacencyMatrix, sizeof(int) * ATSPproblemSize * ATSPproblemSize, hipMemcpyHostToDevice));


    // allocate pheromone matrix on device


    // allocate ant histories on matrix


    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result




    // free all used memory
    cudaHandleError(hipFree(device_STSPAdjacencyMatrix));
    cudaHandleError(hipFree(device_ATSPAdjacencyMatrix));

    free(STSPAdjacencyMatrix);
    free(ATSPAdjacencyMatrix);


}