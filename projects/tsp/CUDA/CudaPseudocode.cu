
#include <string>
#include <string.h>
using namespace std;


void getSTSPAdjacencyMatrix(double* matrix, string location, int problemSize){}

void getATSPAdjacencyMatrix(int* matrix, string location, int nullKey){}

void cudaHandleError(hipError_t error) {
    if(error != hipSuccess){
        cout << "Failed to perform device operation: " << hipGetErrorString(error);
    }
}

int main(){
    

    // for a given problem size
    int STSPproblemSize = 1400; // number of cities
    int ATSPproblemSize = 65; // number of cites

    // and data in local file at a given location
    string STSPLocation = "fl1400.tsp";
    string ATSPLocation = "ftv64.atsp";

    // and possibly some null key for data integrity
    int nullKey = 100000000;

    // populate an adjacency matrix of the problem
    double* STSPAdjacencyMatrix = (double*) malloc(sizeof(double) * STSPproblemSize * STSPproblemSize);
    int* ATSPAdjacencyMatrix = (int*) malloc(sizeof(int) * ATSPproblemSize * ATSPproblemSize);

    
    getSTSPAdjacencyMatrix(STSPAdjacencyMatrix, STSPLocation, STSPproblemSize);
    getATSPAdjacencyMatrix(ATSPAdjacencyMatrix, ATSPLocation, nullKey);


    // create coppies of the problems on the device
    double* device_STSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_STSPAdjacencyMatrix, sizeof(STSPAdjacencyMatrix)));
    cudaHandleError(hipMemcpy(device_STSPAdjacencyMatrix, STSPAdjacencyMatrix, hipMemcpyHostToDevice));

    int* device_ATSPAdjacencyMatrix;
    cudaHandleError(hipMalloc(&device_ATSPAdjacencyMatrix, sizeof(ATSPAdjacencyMatrix)));
    cudaHandleError((device_ATSPAdjacencyMatrix, ATSPAdjacencyMatrix, hipMemcpyHostToDevice));


    // allocate pheromone matrix on device


    // allocate ant histories on matrix


    // invoke kernel

    // check for kernel errors (immediately after kernel execution)



    // get ant histories and find best result




    // free all used memory

    cudaHandleError(hipFree(device_STSPAdjacencyMatrix));
    cudaHandleError(hipFree(device_ATSPAdjacencyMatrix));

    free(STSPAdjacencyMatrix);
    free(ATSPAdjacencyMatrix);


}